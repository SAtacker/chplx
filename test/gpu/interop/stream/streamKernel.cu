#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__global__ void vector_add_kernel(float *a, float *b, float *c, float alpha, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    a[i] = b[i] + alpha * c[i];
}

void checkForCudaDevices() {
    int nDevices;
    hipError_t errCode = hipGetDeviceCount(&nDevices);
    if(errCode != hipSuccess) {
        printf("Error determining number of devices\n");
        exit(1);
    }
    printf("Number of Cuda devices: %i\n", nDevices);
}

void checkCudaError() {
     hipError_t err = hipGetLastError();
     if ( err != hipSuccess ) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));       
     }
}

extern "C" void vector_add_cuda(float *a, float *b, float *c, float alpha, int n) {
    float *d_a, *d_b, *d_c;

    // Allocate device memory and transfer data from host to device
    hipMalloc((void**)&d_a, sizeof(float) * n);
    hipMalloc((void**)&d_b, sizeof(float) * n);
    hipMalloc((void**)&d_c, sizeof(float) * n);

    hipMemcpy(d_a, a, sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, sizeof(float) * n, hipMemcpyHostToDevice);

    // Execute the kernel and copy memory from device back to host
    checkCudaError();
    vector_add_kernel<<<1,n>>>(d_a, d_b, d_c, alpha, n);
    checkCudaError();
    hipMemcpy(a, d_a, sizeof(float) * n, hipMemcpyDeviceToHost);

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
