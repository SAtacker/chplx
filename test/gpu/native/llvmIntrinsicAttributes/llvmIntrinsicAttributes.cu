
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void sample_kernel() {
    int i = threadIdx.x;
    printf("thread %d\n", i);
}

int main() {
  sample_kernel<<<1,1>>>();
}
